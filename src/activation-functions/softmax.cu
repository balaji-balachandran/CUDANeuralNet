#include "hip/hip_runtime.h"
#include "softmax.h"

float* Softmax::Forward(float* input_data){
    input_ = input_data;
    float* layer_output = is_parallelized_ ? ParallelizedForward(input_data) : 
                                             SequentialForward(input_data);

    return layer_output;
}

float* Softmax::SequentialForward(float* input_data){
    float* layer_output = new float[output_size_];
    float normalizer = 0;
    for(size_t i = 0; i < input_size_; i++){
        normalizer += std::exp(input_data[i]);
    }

    for(size_t i = 0; i < input_size_; i++){
        layer_output[i] = std::exp(input_data[i]) / normalizer;
    }
    
    return layer_output;
}

float* Softmax::ParallelizedForward(float* input_data){
    return nullptr;
}

float* Softmax::Backward(float* d_error_d_output){
    float* d_error_d_input = is_parallelized_ ? ParallelizedBackward(d_error_d_output) : 
        SequentialBackward(d_error_d_output);
    
    return d_error_d_input;
}

float* Softmax::SequentialBackward(float* d_error_d_output){
    float* d_error_d_input = new float[input_size_];
    
    float normalizer = 0;
    for(size_t i = 0; i < input_size_; i++){
        normalizer += std::exp(input_[i]);
    }

    for(size_t i = 0; i < input_size_; i++){
        float softmax_input = std::exp(input_[i]) / normalizer;
        d_error_d_input[i] = d_error_d_output[i] * softmax_input * (1 - softmax_input);
    }

    return d_error_d_input;
}

float* Softmax::ParallelizedBackward(float* d_error_d_output){
    return nullptr;
}